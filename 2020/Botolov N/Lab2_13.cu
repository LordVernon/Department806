#include "hip/hip_runtime.h"

#include <ctime>
#include<stdio.h>
#include "iostream"

using namespace std;
#define N 900

__global__ void armNumbWithCuda(int* a)
{
    int idx = threadIdx.x;
    int hund, dec, ones;
    hund = a[idx] / 100;
    dec = a[idx] / 10 % 10;
    ones = a[idx] % 10;
    int r = hund * hund * hund + dec * dec * dec + ones * ones * ones;
    if (r == a[idx])
    {
        printf("%d  ", a[idx]);
    }
}

void createArr(int* a)
{
    int j = 100;
    for (int i = 0; i < N; i++)
    {
        a[i] = j;
        j++;
    }
}

int main()
{
    int* a = new int[N];
    int* dev_a;
    float gpuTime = 0.0;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipError_t cudaStatus;
    createArr(a);

    cudaStatus = hipSetDevice(0);
    cudaStatus = hipMalloc((void**)&dev_a, N * sizeof(int));
    cudaStatus = hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);

    cout << "Armstrong numbers on GPU: ";
    hipEventRecord(start, 0);
    armNumbWithCuda <<<1, N>>> (dev_a);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&gpuTime, start, end);
    cout << "\nGPU compute time: "<< gpuTime << " ms\n";

    hipEventDestroy(start);
    hipEventDestroy(end);
    hipFree(dev_a);

	cout << "Armstrong numbers on CPU: ";
    clock_t begin = clock();
    int hund, dec, ones;
	for (int i = 0; i < N; i++)
	{
        hund = a[i] / 100;
        dec = a[i] / 10 % 10;
        ones = a[i] % 10;
        //int r = hund * hund * hund + dec * dec * dec + ones * ones * ones;
        int r = pow(hund, 3) + pow(dec, 3) + pow(ones, 3);
		if (r == a[i])
        {
            printf("%d  ", a[i]);
        }
	}
    clock_t cend1 = clock();
    double cpuTime = (double)(cend1 - begin) / CLOCKS_PER_SEC * 1000;

    printf("\nCPU compute time: %.4f ms\n", cpuTime);
    return 0;
}
